
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/cudnn_ndconv_layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

__global__ void sync_ndconv_groups() { }

template <typename Dtype>
void CudnnNdConvolutionLayer<Dtype>::Forward_gpu(
  const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  
  #if CUDNN_VERSION_MIN(8, 0, 0)  //0
  int RetCnt;
  bool found_conv_algorithm;
  size_t free_memory, total_memory;
  cudnnConvolutionFwdAlgoPerf_t     fwd_algo_pref_[4];
  //cudnnConvolutionBwdDataAlgoPerf_t bwd_data_algo_pref_[4];

  //get memory sizes
  cudaMemGetInfo(&free_memory, &total_memory);
  #endif

  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    const Dtype* weight = this->blobs_[0]->gpu_data();

    size_t workspace_limit_bytes = this->channels_*sizeof(int);
    for (int j = 0; j < this->kernel_shape_.size(); ++j) {
      workspace_limit_bytes *= kernel_shape_[j];
    }
    ++workspace_limit_bytes;

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      cudnnConvolutionFwdAlgo_t algo;
      #if  CUDNN_VERSION_MIN(8, 0, 0)  // 0
      // choose forward algorithm for filter
      // in forward filter the CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED is not implemented in cuDNN 8
      CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm_v7(handle_[0],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        4,
        &RetCnt,
        fwd_algo_pref_));

      found_conv_algorithm = false;
      for(int n=0;n<RetCnt;n++){
        if (fwd_algo_pref_[n].status == CUDNN_STATUS_SUCCESS &&
            fwd_algo_pref_[n].algo != CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED &&
            fwd_algo_pref_[n].memory < free_memory){
          found_conv_algorithm = true;
          //fwd_algo_[i]                   = fwd_algo_pref_[n].algo;
          //workspace_fwd_sizes_[i]        = fwd_algo_pref_[n].memory;
          algo = fwd_algo_pref_[n].algo;
          break;
        }
      }
      if(!found_conv_algorithm) 
         LOG(ERROR) << "[Forward_gpu()]cuDNN did not return a suitable algorithm for convolution.";
      #else
      // pick the convolution algorithm
      // TODO(shelhamer) this should be done during reshape
      // TODO(shelhamer) the choice of automatic or manual algorithm picking
      // should be exposed in proto
      CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm(handle_[g],
                  bottom_descs_[i],
                  filter_desc_,
                  conv_descs_[i],
                  top_descs_[i],
                  CUDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
                  workspace_limit_bytes,  // memoryLimitInBytes,
                  &algo));
      #endif
      // get minimum size of the workspace needed for the desired algorithm
      size_t workspaceSizeInBytes_temp = 0;

      CUDNN_CHECK(cudnnGetConvolutionForwardWorkspaceSize(handle_[g],
                  bottom_descs_[i],
                  filter_desc_,
                  conv_descs_[i],
                  top_descs_[i],
                  algo,
                  &workspaceSizeInBytes_temp));

      if (workspaceSizeInBytes_temp > workspaceSizeInBytes) {
        workspaceSizeInBytes = workspaceSizeInBytes_temp;
        // free the existing workspace and allocate a new (larger) one
        cudaFree(this->workspace_data_);
        cudaError_t err = cudaMalloc(&(this->workspace_data_),
                          workspaceSizeInBytes);
        if (err != cudaSuccess) {
          // force zero memory path
          algo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
          workspace_data_ = NULL;
          workspaceSizeInBytes = 0;
        }
      }

      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
                  cudnn::dataType<Dtype>::one,
                  bottom_descs_[i], bottom_data + bottom_offset_ * g,
                  filter_desc_, weight + weight_offset_ * g,
                  conv_descs_[i],
                  algo, workspace_data_, workspaceSizeInBytes,
                  cudnn::dataType<Dtype>::zero,
                  top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
#if CUDNN_VERSION_MIN(5, 0, 0)
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
                    cudnn::dataType<Dtype>::one,
                    bias_desc_, bias_data + bias_offset_ * g,
                    cudnn::dataType<Dtype>::one,
                    top_descs_[i], top_data + top_offset_ * g));
#else
        CUDNN_CHECK(cudnnAddTensor_v3(handle_[g],
                    cudnn::dataType<Dtype>::one,
                    bias_desc_, bias_data + bias_offset_ * g,
                    cudnn::dataType<Dtype>::one,
                    top_descs_[i], top_data + top_offset_ * g));
#endif
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_ndconv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CudnnNdConvolutionLayer<Dtype>::Backward_gpu(
  const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down,
  const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
                    cudnn::dataType<Dtype>::one,
                    top_descs_[i],  top_diff + top_offset_ * g,
                    cudnn::dataType<Dtype>::one,
                    bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
#if CUDNN_VERSION_MIN(5, 0, 0)
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(handle_[1*this->group_ +
                    g],
                    cudnn::dataType<Dtype>::one,
                    bottom_descs_[i], bottom_data + bottom_offset_ * g,
                    top_descs_[i],    top_diff + top_offset_ * g,
                    conv_descs_[i],
                    bwd_filter_algo_[i], workspace_[1*this->group_ + g],
                    workspace_bwd_filter_sizes_[i],
                    cudnn::dataType<Dtype>::one,
                    filter_desc_, weight_diff + weight_offset_ * g));
#elif CUDNN_VERSION_MIN(4, 0, 0)
        CUDNN_CHECK(cudnnConvolutionBackwardFilter_v2(handle_[1*this->group_ +
                    g],
                    cudnn::dataType<Dtype>::one,
                    bottom_descs_[i], bottom_data + bottom_offset_ * g,
                    top_descs_[i],    top_diff + top_offset_ * g,
                    conv_descs_[i],
                    cudnn::dataType<Dtype>::one,
                    filter_desc_, weight_diff + weight_offset_ * g));
#else
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(handle_[1*this->group_ +
                    g],
                    cudnn::dataType<Dtype>::one,
                    bottom_descs_[i], bottom_data + bottom_offset_ * g,
                    top_descs_[i],    top_diff + top_offset_ * g,
                    conv_descs_[i],
                    cudnn::dataType<Dtype>::one,
                    filter_desc_, weight_diff + weight_offset_ * g));
#endif
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
#if CUDNN_VERSION_MIN(5, 0, 0)
        CUDNN_CHECK(cudnnConvolutionBackwardData(handle_[2*this->group_ + g],
                    cudnn::dataType<Dtype>::one,
                    filter_desc_, weight + weight_offset_ * g,
                    top_descs_[i], top_diff + top_offset_ * g,
                    conv_descs_[i],
                    bwd_data_algo_[i], workspace_[1*this->group_ + g],
                    workspace_bwd_data_sizes_[i],
                    cudnn::dataType<Dtype>::zero,
                    bottom_descs_[i], bottom_diff + bottom_offset_ * g));
#elif CUDNN_VERSION_MIN(4, 0, 0)
        CUDNN_CHECK(cudnnConvolutionBackwardData_v2(handle_[2*this->group_ + g],
                    cudnn::dataType<Dtype>::one,
                    filter_desc_, weight + weight_offset_ * g,
                    top_descs_[i], top_diff + top_offset_ * g,
                    conv_descs_[i],
                    cudnn::dataType<Dtype>::zero,
                    bottom_descs_[i], bottom_diff + bottom_offset_ * g));
#else
        CUDNN_CHECK(cudnnConvolutionBackwardData(handle_[2*this->group_ + g],
                    cudnn::dataType<Dtype>::one,
                    filter_desc_, weight + weight_offset_ * g,
                    top_descs_[i], top_diff + top_offset_ * g,
                    conv_descs_[i],
                    cudnn::dataType<Dtype>::zero,
                    bottom_descs_[i], bottom_diff + bottom_offset_ * g));
#endif
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_ndconv_groups<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CudnnNdConvolutionLayer);

}  // namespace caffe
#endif
